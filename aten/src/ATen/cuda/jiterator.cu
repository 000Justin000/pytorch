#include "hip/hip_runtime.h"
#include <ATen/jit_macros.h>

#if AT_USE_JITERATOR()

#include <c10/util/variant.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/jiterator.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/jit_utils.h>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/JitLoops.cuh>

#include <iostream>
#include <utility>

namespace at {
namespace native {

#define AT_FOR_8_INPUTS(_)  \
  _(0)                      \
  _(1)                      \
  _(2)                      \
  _(3)                      \
  _(4)                      \
  _(5)                      \
  _(6)                      \
  _(7)

c10::SmallVector<std::string> get_extra_args_typenames(const std::vector<at::Scalar>& extra_args) {
  c10::SmallVector<std::string> args_typenames(extra_args.size());
  for (auto i = 0; i < extra_args.size(); ++i) {
    args_typenames[i] = at::cuda::jit::typeName(extra_args[i].type());
  }
  return args_typenames;
}

int can_vectorize_up_to(at::ScalarType type, char* pointer) {
  switch(type) {
#define DEFINE_CASE(ctype, scalartype)                                   \
    case ScalarType::scalartype : return memory::can_vectorize_up_to<ctype>(pointer);

    AT_FORALL_SCALAR_TYPES_WITH_COMPLEX(DEFINE_CASE)
#undef DEFINE_CASE

    default: TORCH_INTERNAL_ASSERT(false, "Unrecognized ScalarType: ", type);
  }
}

// jitted version of the above
// See Note [Jiterator], this relies on the assumptions enumerated there
int jitted_can_vectorize_up_to(const TensorIteratorBase& iter) {
  const at::ScalarType common_dtype = iter.common_dtype();
  const at::ScalarType result_dtype = common_dtype;

  // Deals with output
  int result = can_vectorize_up_to(result_dtype, static_cast<char*>(iter.data_ptr(0)));

  // Incorporates input(s)
  for (auto i = 1; i < iter.ntensors(); ++i) {
    result = std::min<int>(result, can_vectorize_up_to(common_dtype, static_cast<char*>(iter.data_ptr(i))));
  }

  return result;
}

template<int N>
static std::unique_ptr<OffsetCalculator<N>> make_unique_input_offset_calculator(const TensorIteratorBase& iter) {
  // array size can not be 0, this happens when N == 0
  constexpr int array_size = std::max<int>(N, 1);
  TORCH_INTERNAL_ASSERT(N == iter.ntensors() - iter.noutputs());
  std::array<const int64_t*, array_size> strides;
  int64_t element_sizes[array_size];
  for (int i = 0; i < N; i++) {
    strides[i] = iter.strides(i + iter.noutputs()).data();
    element_sizes[i] = iter.element_size(i + iter.noutputs());
  }
  return std::make_unique<OffsetCalculator<N>>(iter.ndim(), iter.shape().data(), strides.data(), element_sizes);
}

template<int ...Is>
auto OffsetCalculatorType_List_Impl(std::integer_sequence<int, Is...>) -> c10::variant<std::unique_ptr<OffsetCalculator<Is>>...>;

template<int N>
using OffsetCalculatorType_List = decltype(OffsetCalculatorType_List_Impl(std::make_integer_sequence<int, N>{}));


struct OffsetCalculatorVariant {
  using OffsetCalculatorTypes = OffsetCalculatorType_List<8>;

  OffsetCalculatorVariant(const TensorIteratorBase& iter) {
    int N = iter.ninputs();
    switch(N) {
#define DEFINE_CASE(index)        \
      case index : v = make_unique_input_offset_calculator<index>(iter); break;

      AT_FOR_8_INPUTS(DEFINE_CASE)
#undef DEFINE_CASE
      default:
        TORCH_CHECK(false, "OffsetCalculatorVariant not implemented for ninputs = ", N);
    }
  }

  void* data_ptr() {
    return c10::visit([](auto & v){ return static_cast<void*>(v.get()); }, v);
  }

 private:
  OffsetCalculatorTypes v;
};

template<int ...Is>
auto ArrayType_List_Impl(std::integer_sequence<int, Is...>) -> c10::variant<at::detail::Array<char*, Is + 2>...>;

template<int N>
using ArrayType_List = decltype(ArrayType_List_Impl(std::make_integer_sequence<int, N>{}));

struct ArrayVariant {
  // notice: This would produce c10::variant<at::detail::Array<char*, 2...10>>
  using ArrayTypes = ArrayType_List<8>;

  ArrayVariant(const TensorIteratorBase& iter) {
    int N = iter.ntensors();
    // jitted kernels must have at least 1 input and 1 output
    switch(N) {
#define DEFINE_CASE(index)      \
      case index + 2: array = at::detail::Array<char*, index + 2>{}; break;

      AT_FOR_8_INPUTS(DEFINE_CASE)
#undef DEFINE_CASE

      default:
        TORCH_CHECK(false, "ArrayVariant not implemented for ninputs = ", N);
    }

    c10::visit([&](auto& a) {
      for (auto i = 0; i < N; ++i) {
        a[i] = (char*)iter.data_ptr(i);
      }
    }, array);
  }

  void* data_ptr() {
    return c10::visit([](auto & a){ return static_cast<void*>(&a); }, array);
  }

private:
  ArrayTypes array;
};

template<int ...Is>
auto TrivialOffsetCalculator_List_Impl(std::integer_sequence<int, Is...>) -> c10::variant<TrivialOffsetCalculator<Is>...>;

template<int N>
using TrivialOffsetCalculator_List = decltype(TrivialOffsetCalculator_List_Impl(std::make_integer_sequence<int, N>{}));

struct TrivialOffsetCalculatorVariant {
  using TrivialOffsetCalculatorTypes = TrivialOffsetCalculator_List<8>;

  TrivialOffsetCalculatorVariant(int arity) {
    switch(arity) {
#define DEFINE_CASE(index)      \
      case index: v = TrivialOffsetCalculator<index>(); break;

      AT_FOR_8_INPUTS(DEFINE_CASE)
#undef DEFINE_CASE

      default:
        TORCH_CHECK(false, "TrivialOffsetCalculatorVariant not implemented for ninputs = ", arity);
    }
  }

  void* data_ptr() {
    return c10::visit([](auto & v){ return static_cast<void*>(&v); }, v);
  }

private:
  TrivialOffsetCalculatorTypes v;
};

template<int ...Is>
auto LoadWithCastPtr_List_Impl(std::integer_sequence<int, Is...>) -> c10::variant<std::unique_ptr<memory::LoadWithCast<Is>>...>;

template<int N>
using LoadWithCastPtr_List = decltype(LoadWithCastPtr_List_Impl(std::make_integer_sequence<int, N>{}));

struct LoadWithCastVariant {
  using LoadWithCastPtr = LoadWithCastPtr_List<8>;

  LoadWithCastVariant(const TensorIteratorBase& iter) {
    int arity = iter.ninputs();
    switch(arity) {
#define DEFINE_CASE(index)      \
      case index: v = std::make_unique<memory::LoadWithCast<index>>(iter); break;

      AT_FOR_8_INPUTS(DEFINE_CASE)
#undef DEFINE_CASE

      default:
        TORCH_CHECK(false, "make_input_offset_calculator not implemented for ninputs = ", arity);
    }
  }

  void* data_ptr() {
    return c10::visit([](auto & v){ return static_cast<void*>(v.get()); }, v);
  }

private:
  LoadWithCastPtr v;
};

static inline void launch_jitted_vectorized_kernel(
  const std::string& name, TensorIteratorBase& iter,
  DeviceIndex dev_idx, int64_t N, const std::string& f, void* data_ptr,
  const std::vector<at::Scalar>& extra_args) {
  TORCH_INTERNAL_ASSERT(N > 0 && N <= std::numeric_limits<int32_t>::max());
  // N is still int64_t for the computation, but it's always safe to cast result to int
  const uint32_t grid = (N + block_work_size() - 1) / block_work_size();

  // TODO: double check here, only seeing vec_size = 4, even for double
  const int vec_size = jitted_can_vectorize_up_to(iter);
  bool vectorized = vec_size > 1;

  // Different kernels are compiled depending on what we're vectorizing up to (1, 2 or 4 elements)
  //   fn_ptr is set to the appropriate function based on the vec size and GPU used
  // TODO: Memory use can probably be optimized by re-using kernels across GPUs with
  //   the same compute capability

  int nTensors =  iter.ntensors();
  const at::ScalarType common_dtype = iter.common_dtype();
  std::string f_inputs_type_str = at::cuda::jit::typeName(common_dtype);
  std::string compute_type_str = at::cuda::jit::typeName(toOpMathType(common_dtype));
  std::string result_type_str = at::cuda::jit::typeName(common_dtype);
  c10::SmallVector<std::string> extra_args_types = get_extra_args_typenames(extra_args);

  // The cache key includes all the parameters to generate_code + vec_size + dev_idx
  std::stringstream ss;
  ss << nTensors << f << name;
  ss << f_inputs_type_str << compute_type_str << result_type_str;
  ss << static_cast<int>(at::cuda::jit::BinaryFuncVariant::NoScalar);
  ss << extra_args_types;
  ss << vec_size;
// DeviceIndex, e.g. int8_t, is not treated as a number by the stream, cast to int as a workaround
  ss << static_cast<int>(dev_idx);
  const std::string cache_key = ss.str();

  static std::mutex _jiterator_mutex;
  static std::unordered_map<std::string, at::cuda::jit::NvrtcFunction> fns;
  at::cuda::jit::NvrtcFunction* fn_ptr = &fns[cache_key];

  if (!fn_ptr->function) {
    const std::lock_guard<std::mutex> lock{_jiterator_mutex};
    if (!fn_ptr->function) { // cache miss!
      // Generates program
      auto code = at::cuda::jit::generate_code(nTensors, f, name,
                                               f_inputs_type_str, compute_type_str, result_type_str,
                                               /*contiguous=*/true, /*dynamic_casting=*/false,
                                               at::cuda::jit::BinaryFuncVariant::NoScalar,
                                               extra_args_types,
                                               vectorized, vec_size);
      std::string kernel_name = vectorized ? name + "_vectorized" + std::to_string(vec_size) : name;

      // Acquires the program
      *fn_ptr = at::cuda::jit::jit_pwise_function(code, kernel_name);
    }
  }

  // size of `extra_args` is unknown at compile-time
  auto extra_args_size = extra_args.size();

  float scalar_val = 0;

  if (vectorized) {
    // pack args for kernel launch
    constexpr int kernel_args = 3;
    auto args = std::make_unique<void*[]>(kernel_args + extra_args_size);
    args[0] = static_cast<void*>(&N);
    args[1] = data_ptr;
    args[2] = static_cast<void*>(&scalar_val);

    for (const auto i : c10::irange(extra_args_size)) {
      // since 3 slots are already filled in `args`
      args[i + 3] = const_cast<void*>(extra_args[i].data_ptr());
    }
    at::cuda::jit::launch_jitted_pwise_function(*fn_ptr, args.get(), {grid, 1u, 1u}, {num_threads(), 1u, 1u});
  } else {
    TrivialOffsetCalculatorVariant input_offset_calculator(iter.ninputs());
    void* ic_ptr = input_offset_calculator.data_ptr();
    auto oc = TrivialOffsetCalculator<1>();
    auto l = memory::LoadWithoutCast();
    auto s = memory::StoreWithoutCast();

    // pack args for kernel launch
    constexpr int kernel_args = 7;
    auto args = std::make_unique<void*[]>(kernel_args + extra_args_size);
    args[0] = static_cast<void*>(&N);
    args[1] = data_ptr;
    args[2] = ic_ptr;
    args[3] = static_cast<void*>(&oc);
    args[4] = static_cast<void*>(&l);
    args[5] = static_cast<void*>(&s);
    args[6] = static_cast<void*>(&scalar_val);

    for (const auto i : c10::irange(extra_args_size)) {
      // since 7 slots are already filled in `args`
      args[i + 7] = const_cast<void*>(extra_args[i].data_ptr());
    }

    at::cuda::jit::launch_jitted_pwise_function(*fn_ptr, args.get(), {grid, 1u, 1u}, {num_threads(), 1u, 1u});
  }
}

static inline void launch_jitted_unrolled_kernel(
  const std::string& name, TensorIteratorBase& iter,
  DeviceIndex dev_idx, int64_t N, const std::string& f, void* data_ptr,
  void* ic_ptr, void* oc_ptr, void* l_ptr, void* s_ptr, bool contiguous, bool dynamic_casting,
  const std::vector<at::Scalar>& extra_args) {

  TORCH_INTERNAL_ASSERT(N > 0 && N <= std::numeric_limits<int32_t>::max());
  //casting result to int is always safe, intermediate is int64 and won't overflow
  const uint32_t grid = (N + block_work_size() - 1) / block_work_size();

  int nTensors = iter.ntensors();
  const at::ScalarType common_dtype = iter.common_dtype();
  std::string f_inputs_type_str = at::cuda::jit::typeName(common_dtype);
  std::string compute_type_str = at::cuda::jit::typeName(toOpMathType(common_dtype));
  std::string result_type_str = at::cuda::jit::typeName(common_dtype);
  c10::SmallVector<std::string> extra_args_types = get_extra_args_typenames(extra_args);

  // The cache key includes all the parameters to generate_code + dev_idx
  std::stringstream ss;
  ss << nTensors << f << name;
  ss << f_inputs_type_str << compute_type_str << result_type_str;
  ss << contiguous << dynamic_casting;
  ss << static_cast<int>(at::cuda::jit::BinaryFuncVariant::NoScalar);
  ss << extra_args_types;
  ss << dev_idx;
  const std::string cache_key = ss.str();

  static std::mutex _jiterator_mutex;
  static std::unordered_map<std::string, at::cuda::jit::NvrtcFunction> fns;

  at::cuda::jit::NvrtcFunction* fn_ptr = &fns[cache_key];
  if (!fn_ptr->function) {
    const std::lock_guard<std::mutex> lock{_jiterator_mutex};
    if (!fn_ptr->function) {
      auto code = at::cuda::jit::generate_code(nTensors, f, name,
                                               f_inputs_type_str, compute_type_str, result_type_str,
                                               contiguous, dynamic_casting,
                                               at::cuda::jit::BinaryFuncVariant::NoScalar,
                                               extra_args_types);
      *fn_ptr = at::cuda::jit::jit_pwise_function(code, name);
    }
  }

  float scalar_val = 0;

  // pack args for kernel launch
  constexpr int kernel_args = 7;
  auto extra_args_size = extra_args.size();
  auto args = std::make_unique<void*[]>(kernel_args + extra_args_size);
  args[0] = static_cast<void*>(&N);
  args[1] = data_ptr;
  args[2] = ic_ptr;
  args[3] = oc_ptr;
  args[4] = l_ptr;
  args[5] = s_ptr;
  args[6] = static_cast<void*>(&scalar_val);

  for (const auto i : c10::irange(extra_args_size)) {
    // since 7 slots are already filled in `args`
    args[i + 7] = const_cast<void*>(extra_args[i].data_ptr());
  }
  at::cuda::jit::launch_jitted_pwise_function(*fn_ptr, args.get(), {grid, 1u, 1u}, {num_threads(), 1u, 1u});
}

void jitted_gpu_kernel_dynamic_impl(
    const std::string& kernel_name,
    TensorIteratorBase& iter,
    const std::string& f,
    const bool dynamic_casting,
    const std::vector<at::Scalar>& extra_args) {

  TORCH_INTERNAL_ASSERT(iter.can_use_32bit_indexing());
  TORCH_INTERNAL_ASSERT(iter.noutputs() == 1);

  // TODO: assuming supported ninputs <=8, with only one output
  TORCH_INTERNAL_ASSERT(iter.ninputs() <= 8);

  ArrayVariant data(iter);
  void* data_ptr = data.data_ptr();

  int64_t numel = iter.numel();
  bool contiguous = iter.is_contiguous();

  // Decides which of 4 kernel types to launch
  // Variations are:
  //   - Case 1: no dynamic casting and contiguous
  //   - Case 2: no dynamic casting and noncontiguous
  //   - Case 3: dynamic casting and contiguous
  //   - Case 4: dynamic casting and noncontiguous
  // These cases align with the non-jitted CUDALoops.cuh cases in gpu_kernel_impl

  if (!dynamic_casting) {
    if (contiguous) {
      // Case 1: no dynamic casting and contiguous
      launch_jitted_vectorized_kernel(kernel_name, iter,
         iter.device().index(), numel, f, data_ptr, extra_args);
      return;
    }

    // Case 2: no dynamic casting and noncontiguous
    OffsetCalculatorVariant input_offset_calculator(iter);
    void* ic_ptr = input_offset_calculator.data_ptr();
    auto output_offset_calculator = make_output_offset_calculator(iter);
    void* oc_ptr = static_cast<void*>(&output_offset_calculator);

    auto loader = memory::LoadWithoutCast();
    auto storer = memory::StoreWithoutCast();
    void* l_ptr = static_cast<void*>(&loader);
    void* s_ptr = static_cast<void*>(&storer);

    launch_jitted_unrolled_kernel(
      kernel_name, iter, iter.device().index(), numel, f, data_ptr,
      ic_ptr, oc_ptr, l_ptr, s_ptr, contiguous, dynamic_casting, extra_args);

    return;
  }

  // Cases 3 and 4 are handled below
  // Both require construction of a storer (this asserts 1 output) and one or more loaders

  // Creates load casts from inputs (note offset indexing into the iterators 1...n tensors)
  LoadWithCastVariant loader(iter);
  void* l_ptr = loader.data_ptr();

  // Creates store cast to output (the zeroth tensor in TensorIterator)
  auto storer = memory::StoreWithCast(iter.dtype(0));
  void* s_ptr = static_cast<void*>(&storer);

  if (contiguous) {
    // Case 3: dynamic casting and contiguous
    TrivialOffsetCalculatorVariant input_offset_calculator(iter.ninputs());
    void* ic_ptr = input_offset_calculator.data_ptr();

    auto output_offset_calculator = TrivialOffsetCalculator<1>();
    void* oc_ptr = static_cast<void*>(&output_offset_calculator);

    launch_jitted_unrolled_kernel(
      kernel_name, iter, iter.device().index(), numel, f, data_ptr,
      ic_ptr, oc_ptr, l_ptr, s_ptr, contiguous, dynamic_casting, extra_args);
    return;
  }

  // Case 4: dynamic casting and noncontiguous
  OffsetCalculatorVariant input_offset_calculator(iter);
  void* ic_ptr = input_offset_calculator.data_ptr();

  auto output_offset_calculator = make_output_offset_calculator(iter);
  void* oc_ptr = static_cast<void*>(&output_offset_calculator);

  launch_jitted_unrolled_kernel(
      kernel_name, iter, iter.device().index(), numel, f, data_ptr,
      ic_ptr, oc_ptr, l_ptr, s_ptr, contiguous, dynamic_casting, extra_args);
}

void jitted_gpu_kernel_dynamic(
    const std::string& kernel_name,
    TensorIteratorBase& iter,
    const std::string& f,
    const std::vector<at::Scalar>& extra_args) {

  // TODO: much of preamble is common to both jitted_gpu_kernel and gpu_kernel
  //   Maybe it could be refactored?
  for (int arg = 0; arg < iter.ntensors(); arg++) {
    TORCH_INTERNAL_ASSERT(
      iter.device(arg).is_cuda(),
      "argument ", arg, ": expected a CUDA device but found ", iter.device(arg));
  }

  if (iter.numel() == 0) {
    return;
  }

  if (!iter.can_use_32bit_indexing()) {
    for (auto& sub_iter : iter.with_32bit_indexing()) {
      jitted_gpu_kernel_dynamic(kernel_name, sub_iter, f, extra_args);
    }
    return;
  }

  // Computes if dynamic casting is needed
  // Dynamic casting is needed if an input's or output's dtype differs from the common dtype
  // TODO: double check! this is different from jitted_gpu_kernel's logic
  bool needs_dynamic_casting = false;
  const at::ScalarType common_dtype = iter.common_dtype();
  for (auto i = 0; i < iter.ntensors(); ++i) {
    if (iter.dtype(i) != common_dtype) {
      needs_dynamic_casting = true;
      break;
    }
  }

  jitted_gpu_kernel_dynamic_impl(kernel_name, iter, f, needs_dynamic_casting, extra_args);
}

} // namespace native



namespace cuda {

at::Tensor CompileKernel(
  const std::string& op_string,
  const std::string& kernel_name,
  const std::vector<at::Tensor>& tensors,
  const std::vector<at::Scalar>& extra_args) {

  Tensor output;
  // TODO: double check if any other flags needs to be set
  TensorIteratorConfig config;
  config
    .set_check_mem_overlap(true)
    .allow_cpu_scalars(false)
    .promote_inputs_to_common_dtype(true)
    .cast_common_dtype_to_outputs(true)
    .enforce_safe_casting_to_output(true)
    .check_all_same_device(true)
    // TODO:  add_output or add_owned_output
    .add_owned_output(output);
  for (const auto& t: tensors){
    config.add_input(t);
  }
  TensorIterator iter = config.build();

  CUDAGuard guard(iter.device());
  at::native::jitted_gpu_kernel_dynamic(kernel_name, iter, op_string, extra_args);

  return iter.output();
}

}} // namespace at::cuda

#endif // AT_USE_JITERATOR()
